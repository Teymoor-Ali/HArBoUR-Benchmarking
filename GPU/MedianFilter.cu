#include <opencv2/opencv.hpp>
#include <opencv2/cudafilters.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    // Load the image
    cv::Mat src = cv::imread("input_image.jpg", cv::IMREAD_COLOR);
    if (src.empty()) {
        std::cerr << "Error: Unable to load image!" << std::endl;
        return -1;
    }

    // Convert to GPU Mat
    cv::cuda::GpuMat d_src(src);
    cv::cuda::GpuMat d_dst;

    // Create a median filter
    cv::Ptr<cv::cuda::Filter> medianFilter = cv::cuda::createMedianFilter(d_src.type(), 5);

    // Timing the operation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    medianFilter->apply(d_src, d_dst);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Median Filter Time: " << milliseconds << " ms" << std::endl;

    // Download the result to host
    cv::Mat result;
    d_dst.download(result);

    // Save the result image
    cv::imwrite("output_median_filter.jpg", result);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
